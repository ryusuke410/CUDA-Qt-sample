#include "hip/hip_runtime.h"
﻿#include "vectadd.cuh"

namespace vectadd {

__global__
void vectAddEqCUDA(int count, float *vec0, const float *vec1);

}  // namespace vectadd

namespace vectadd {

void vectAdd(int count, const float *vec0, const float *vec1, float *added) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);

  const int blockSize = deviceProp.maxThreadsPerBlock;
  const int blockCount = (count - 1) / blockSize + 1;

  float *d_vec0;
  float *d_vec1;

  hipMalloc(&d_vec0, count * sizeof(float));
  hipMalloc(&d_vec1, count * sizeof(float));

  hipMemcpy(d_vec0, vec0, count * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vec1, vec1, count * sizeof(float), hipMemcpyHostToDevice);

  vectAddEqCUDA<<<blockCount, blockSize>>>(count, d_vec0, d_vec1);

  hipMemcpy(added, d_vec0, count * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_vec0);
  hipFree(d_vec1);
}

__global__
void vectAddEqCUDA(int count, float *vec0, const float *vec1)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < count; i += stride) {
    vec0[i] += vec1[i];
  }
}

}  // namespace vectadd
